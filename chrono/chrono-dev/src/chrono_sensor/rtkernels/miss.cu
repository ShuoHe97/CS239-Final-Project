#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for coloring upon ray not intersecting anything
//
// =============================================================================

#include <hip/hip_math_constants.h>
#include <optixu/optixu_aabb.h>
#include "chrono_sensor/rtkernels/ray_utils.h"

using namespace optix;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
// rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, scene_epsilon, , );
// rtDeclareVariable(float, max_scene_distance, , );
// rtDeclareVariable(rtObject, root_node, , );
rtDeclareVariable(float3, default_color, , );
rtDeclareVariable(float, default_depth, , );

// camera parameters
// rtDeclareVariable(float3, c_pos, , );      // camera position
// rtDeclareVariable(float3, c_up, , );       // camera up vector
// rtDeclareVariable(float3, c_forward, , );  // camera forward vector
// rtDeclareVariable(float3, c_left, , );     // camera left vector
// rtDeclareVariable(float, c_hFOV, , );      // camera horizontal field of view
// rtDeclareVariable(float, c_vFOV, , );      // camera vertical field of view

// environment map
rtTextureSampler<float4, 2> environment_map;
rtDeclareVariable(int, has_environment_map, , );

RT_PROGRAM void miss_function() {
    if (prd_radiance.type == DEPTH) {
        prd_radiance.distance = default_depth;
    } else if (prd_radiance.type == LIDAR) {
        prd_radiance.color = make_float3(default_depth, 0.f, 0.f);
    } else if (prd_radiance.type == CAMERA) {
        if (has_environment_map) {
            float theta = atan2f(ray.direction.x, ray.direction.y);
            float phi = asinf(ray.direction.z);
            float tex_x = theta / (2 * M_PIf);
            float tex_y = phi / (M_PIf) + 0.5;

            prd_radiance.color = make_float3(tex2D(environment_map, tex_x, tex_y));
            // prd_radiance.distance = default_depth;
            // if (prd_radiance.first_distance < 1e-3)
            //     prd_radiance.first_distance = default_depth;
            // if (prd_radiance.first_distance < 1e-3)
            //     prd_radiance.first_distance = default_depth;
        } else {
            prd_radiance.color = default_color;
            // prd_radiance.distance = default_depth;
            if (prd_radiance.first_distance < scene_epsilon)
                prd_radiance.first_distance = default_depth;
            // if (prd_radiance.first_distance < 1e-3)
            //     prd_radiance.first_distance = default_depth;
        }
    }
}
