#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for tracing and measureing depth for a typical pinhole camera
//
// =============================================================================

#include <hip/hip_math_constants.h>
#include <optixu/optixu_aabb.h>
#include "chrono_sensor/rtkernels/ray_utils.h"

using namespace optix;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, max_scene_distance, , );
rtDeclareVariable(rtObject, root_node, , );
rtDeclareVariable(float3, default_color, , );
rtDeclareVariable(float, default_depth, , );

// camera parameters
rtDeclareVariable(float3, c_pos, , );      // camera position
rtDeclareVariable(float3, c_up, , );       // camera up vector
rtDeclareVariable(float3, c_forward, , );  // camera forward vector
rtDeclareVariable(float3, c_left, , );     // camera left vector
rtDeclareVariable(float, c_hFOV, , );      // camera horizontal field of view
rtDeclareVariable(float, c_vFOV, , );      // camera vertical field of view

// environment map
rtTextureSampler<float4, 2> environment_map;
rtDeclareVariable(int, has_environment_map, , );

rtBuffer<uchar4, 2> output_buffer;  // byte version

// This kernel is launched once for each pixel in the image
RT_PROGRAM void pinhole_camera() {
    size_t2 screen = output_buffer.size();

    // set the ray direction based on the proportion of image the pixel is located at
    float2 d = (make_float2(launch_index) + make_float2(0.5, 0.5)) / make_float2(screen) * 2.f - 1.f;
    // origin of the camera is  0,0,0 for now
    float3 ray_origin = c_pos;
    float h_factor = c_hFOV / HIP_PI_F * 2.0;
    float v_factor = c_vFOV / HIP_PI_F * 2.0;
    float3 ray_direction = normalize(c_forward - d.x * c_left * h_factor + d.y * c_up * v_factor);

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, max_scene_distance);

    // set the ray pay load
    PerRayData_radiance prd_radiance = make_radiance_data(make_float3(0), 1.f, 0, 0, 0, CAMERA);

    // launch the ray
    rtTrace(root_node, ray, prd_radiance);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] = make_color(prd_radiance.color);
}
