#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for tracing and measureing depth for a typical pinhole camera
//
// =============================================================================

#include <hip/hip_math_constants.h>
#include <optixu/optixu_aabb.h>
#include "chrono_sensor/rtkernels/ray_utils.h"

using namespace optix;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, max_scene_distance, , );
rtDeclareVariable(rtObject, root_node, , );

// camera parameters
rtDeclareVariable(float3, c_pos, , );      // lidar position
rtDeclareVariable(float3, c_up, , );       // lidar up vector
rtDeclareVariable(float3, c_forward, , );  // lidar forward vector
rtDeclareVariable(float3, c_left, , );     // lidar left vector
rtDeclareVariable(float, c_hFOV, , );      // lidar horizontal field of view
rtDeclareVariable(float, c_vFOV, , );      // lidar vertical field of view

rtBuffer<float2, 2> output_buffer;  // byte version

// This kernel is launched once for each pixel in the image
RT_PROGRAM void spherical() {
    size_t2 screen = output_buffer.size();

    // set the ray direction based on the proportion of image the pixel is located at
    float2 d = (make_float2(launch_index) + make_float2(0.5, 0.5)) / make_float2(screen) * 2.f - 1.f;  //[-1,1]

    float theta = d.x * c_hFOV / 2.0;
    float phi = d.y * c_vFOV / 2.0;
    float xy_proj = cos(phi);

    float z = sin(phi);
    float y = xy_proj * sin(theta);
    float x = xy_proj * cos(theta);

    // origin of the camera is  0,0,0 for now
    float3 ray_origin = c_pos;
    float3 ray_direction = normalize(c_forward * x + c_left * y + c_up * z);

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, max_scene_distance);

    // set the ray pay load
    PerRayData_radiance prd_radiance;
    prd_radiance.importance = 1.f;
    prd_radiance.depth = 0;
    prd_radiance.type = LIDAR;

    // launch the ray
    rtTrace(root_node, ray, prd_radiance);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] =
        make_float2(prd_radiance.color.x, prd_radiance.color.y);  // make_color(prd_radiance.color);
}
