#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for box geometries
//
// =============================================================================
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "chrono_sensor/rtkernels/ray_utils.h"
#include "chrono_sensor/scene/lights.h"

using namespace optix;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, transparency, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float, fresnel_exp, , );
rtDeclareVariable(float, fresnel_min, , );
rtDeclareVariable(float, fresnel_max, , );

rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(int, has_texture, , );
rtTextureSampler<float4, 2> Kd_map;

rtDeclareVariable(rtObject, root_node, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, max_scene_distance, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

rtBuffer<PointLight> lights;

RT_PROGRAM void normal_shader() {
    // set the result equal to distance to the intersection
    if (prd_radiance.type == DEPTH) {
        prd_radiance.distance = t_hit;
    } else if (prd_radiance.type == LIDAR) {
    } else if (prd_radiance.type == CAMERA) {
        prd_radiance.color = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)) * 0.5f + 0.5f;
    }
}

RT_PROGRAM void reflective_shader() {
    // set the result equal to distance to the intersection
    if (prd_radiance.type == DEPTH) {
        prd_radiance.distance = t_hit;
        prd_radiance.first_distance = t_hit;
    } else if (prd_radiance.type == LIDAR) {
        prd_radiance.first_distance = t_hit;
        float3 geometric_normal_glob = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
        float3 shading_normal_glob = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
        float3 forward_normal = faceforward(shading_normal_glob, -ray.direction, geometric_normal_glob);

        float intensity = dot(forward_normal, -ray.direction);

        prd_radiance.color = make_float3(t_hit, intensity, 0.f);
    } else if (prd_radiance.type == CAMERA) {
        prd_radiance.first_distance = t_hit;
        float3 geometric_normal_glob = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
        float3 shading_normal_glob = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
        // make sure we have the normal that is facing where the ray came from
        float3 forward_normal = faceforward(shading_normal_glob, -ray.direction, geometric_normal_glob);
        // float3 forward_normal = faceforward(geometric_normal_glob, -ray.direction, geometric_normal_glob);

        float3 resulting_color = make_float3(0.0f);
        //=================
        // ambient color
        //=================

        // get Kd either from color or from texture
        float3 tmp_kd = Kd;
        float3 tmp_ka = Ka;
        // float3 tmp_ka = Kd;
        if (has_texture) {
            tmp_kd = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
            tmp_ka = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
        }

        resulting_color += tmp_ka * ambient_light_color;  // the color is at least equal to ambient color factor

        float3 hit_point = ray.origin + ray.direction * t_hit;  // + 0.01 * forward_normal;
        for (int i = 0; i < lights.size(); i++) {
            PointLight l = lights[i];
            float dist_to_light = length(l.pos - hit_point);
            float3 dir_to_light = normalize(l.pos - hit_point);

            //
            float light_normal_intensity = dot(forward_normal, dir_to_light);

            // 0 if we already know there is a shadow, 1 if we might be able to see the light
            float3 light_attenuation = make_float3(static_cast<float>(light_normal_intensity > 0.f));

            // if we think we can see the light, let's see if we are correct
            if (light_normal_intensity > 0.0f) {
                // float light_attenuation = clamp(((l.max_range - dist_to_light) / l.max_range), 0.0f, 1.0f);
                light_attenuation = make_float3(light_normal_intensity);
                // light_normal_intensity =
                //     light_normal_intensity * clamp(((l.max_range - dist_to_light) / l.max_range), 0.0f, 1.0f);

                // check shadows
                PerRayData_shadow prd_shadow;
                prd_shadow.attenuation = make_float3(1.0f);
                Ray shadow_ray(hit_point, dir_to_light, SHADOW_RAY_TYPE, scene_epsilon, dist_to_light);
                rtTrace(root_node, shadow_ray, prd_shadow);
                light_attenuation = prd_shadow.attenuation;
            }

            // if any of our channels can see the light, let's calculate the contribution
            if (fmaxf(light_attenuation) > 0.0f) {
                // linear light fall off -> TODO: change to fall with 1/r^2 (surface area expansion)

                float3 light_contrib = l.color * light_normal_intensity * light_attenuation *
                                       clamp(((l.max_range - dist_to_light) / l.max_range), 0.0f, 1.0f);

                // light_contrib = light_contrib * light_attenuation;
                // diffuse component with point light falling off

                //=================
                // diffuse color
                //=================
                // resulting_color += tmp_kd * light_normal_intensity * light_contrib;
                resulting_color += tmp_kd * light_contrib;

                // printf("Dot to light: %f\n", light_normal_intensity);

                // specular component
                float3 halfway_vec = normalize(dir_to_light - ray.direction);
                float dot_fn_halfway = dot(forward_normal, halfway_vec);
                if (dot_fn_halfway > 0.0f) {
                    //=================
                    // specular color
                    //=================
                    resulting_color += Ks * light_contrib * pow(dot_fn_halfway, phong_exp);
                }
            }
        }

        // calculate transparency and send ray through material
        float refract_importance = prd_radiance.importance * (1 - transparency);
        if (refract_importance > importance_cutoff && prd_radiance.depth < max_depth) {
            PerRayData_radiance prd_refraction =
                make_radiance_data(make_float3(0), refract_importance, prd_radiance.depth + 1, 0, 0, CAMERA);

            float3 refract_dir;
            refract(refract_dir, ray.direction, forward_normal, 1.f);
            Ray refraction_ray(hit_point, refract_dir, RADIANCE_RAY_TYPE, scene_epsilon, max_scene_distance);
            rtTrace(root_node, refraction_ray, prd_refraction);

            //==============================================
            // composite color from refraction and diffusion
            //==============================================
            resulting_color = transparency * resulting_color + (1 - transparency) * prd_refraction.color;
        }

        float3 reflect_amount = Ks * fresnel_schlick(dot(forward_normal, -ray.direction), fresnel_exp,
                                                     make_float3(fresnel_min), make_float3(fresnel_max));

        float reflect_importance = prd_radiance.importance * luminance(reflect_amount);

        if (reflect_importance > importance_cutoff && prd_radiance.depth < max_depth) {
            // PerRayData_radiance prd_reflection;
            PerRayData_radiance prd_reflection =
                make_radiance_data(make_float3(0), reflect_importance, prd_radiance.depth + 1, 0, 0, CAMERA);

            float3 reflect_dir = reflect(ray.direction, forward_normal);
            Ray reflection_ray(hit_point, reflect_dir, RADIANCE_RAY_TYPE, scene_epsilon, max_scene_distance);

            rtTrace(root_node, reflection_ray, prd_reflection);

            //==============================================
            // composite color from reflection and diffusion
            //==============================================
            resulting_color = (1 - reflect_amount) * resulting_color + reflect_amount * prd_reflection.color;
        }

        prd_radiance.color = resulting_color;
    }
}

RT_PROGRAM void hit_shadow() {
    // if the shadow ray hits anything before reaching the light, light clearly cannot hit this point
    prd_shadow.attenuation = make_float3(0.0f);
    rtTerminateRay();
}
