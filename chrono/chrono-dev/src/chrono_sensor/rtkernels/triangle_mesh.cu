#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for mesh geometries
//
// =============================================================================

#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

#include "chrono_sensor/rtkernels/ray_utils.h"
using namespace optix;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(int, has_texture, , );

rtBuffer<int3> index_buffer;
rtBuffer<float3> vertex_buffer;
rtBuffer<unsigned int> material_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;

RT_PROGRAM void mesh_attributes() {
    const int3 vertex_idx = index_buffer[rtGetPrimitiveIndex()];
    const float3 v0 = vertex_buffer[vertex_idx.x];
    const float3 v1 = vertex_buffer[vertex_idx.y];
    const float3 v2 = vertex_buffer[vertex_idx.z];

    geometric_normal = normalize(cross(v1 - v0, v2 - v0));

    const float2 bary_coord = rtGetTriangleBarycentrics();

    if (normal_buffer.size() == 0) {
        shading_normal = geometric_normal;
    } else {
        shading_normal =
            normalize(normal_buffer[vertex_idx.y] * bary_coord.x + normal_buffer[vertex_idx.z] * bary_coord.y +
                      normal_buffer[vertex_idx.x] * (1.0f - bary_coord.x - bary_coord.y));
    }
    if (texcoord_buffer.size() == 0) {
        texcoord = make_float2(0.0f);
    } else {
        texcoord = texcoord_buffer[vertex_idx.y] * bary_coord.x + texcoord_buffer[vertex_idx.z] * bary_coord.y +
                   texcoord_buffer[vertex_idx.x] * (1.0f - bary_coord.x - bary_coord.y);
    }
}
