#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for box geometries
//
// =============================================================================
#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>

#include "chrono_sensor/rtkernels/ray_utils.h"
using namespace optix;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );
rtDeclareVariable(int, has_texture, , );

static __device__ float3 box_normal(float t, float3 t0, float3 t1) {
    float3 normal_pos = make_float3(t == t0.x ? 1 : 0, t == t0.y ? 1 : 0, t == t0.z ? 1 : 0);
    float3 normal_neg = make_float3(t == t1.x ? 1 : 0, t == t1.y ? 1 : 0, t == t1.z ? 1 : 0);
    return normal_pos - normal_neg;
}

RT_PROGRAM void box_intersect(int) {
    // calculate potential intersections with the box
    float3 t0 = (boxmin - ray.origin) / ray.direction;
    float3 t1 = (boxmax - ray.origin) / ray.direction;
    float3 near = fminf(t0, t1);
    float3 far = fmaxf(t0, t1);
    // dist_near and dist_far are the distances to the potential intsection points
    float dist_near = fmaxf(near);
    float dist_far = fminf(far);

    // check if near is less than far
    if (dist_near <= dist_far) {
        bool check_second = true;
        if (rtPotentialIntersection(dist_near)) {
            shading_normal = geometric_normal = box_normal(dist_near, t0, t1);
            float3 p = ray.origin + dist_near * ray.direction;
            float3 scaled =
                (p - boxmin) / (boxmax - boxmin) -
                make_float3(2 * abs(shading_normal.x), 2 * abs(shading_normal.y), 2 * abs(shading_normal.z));
            float u = scaled.x > -.5f ? scaled.x : scaled.y;
            float v = (scaled.x > -.5f && scaled.y > -.5f) ? scaled.y : scaled.z;
            texcoord = make_float2(u, v);
            if (rtReportIntersection(0))
                check_second = false;
        }
        if (check_second) {
            if (rtPotentialIntersection(dist_far)) {
                shading_normal = geometric_normal = box_normal(dist_far, t0, t1);
                float3 p = ray.origin + dist_far * ray.direction;
                float3 scaled = (p - boxmin) / (boxmax - boxmin) -
                                make_float3(abs(shading_normal.x), abs(shading_normal.y), abs(shading_normal.z));
                float u = scaled.x > -.5f ? abs(scaled.x) : abs(scaled.y);
                float v = scaled.x > -.5f ? abs(scaled.y) : abs(scaled.z);
                texcoord = make_float2(u, v);
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void box_bounds(int, float result[6]) {
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set(boxmin, boxmax);
}
