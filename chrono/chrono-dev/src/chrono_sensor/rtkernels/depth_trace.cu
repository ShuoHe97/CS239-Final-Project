#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// RT kernels for tracing and measureing depth for a typical pinhole camera
//
// =============================================================================

#include <optixu/optixu_aabb.h>
#include "chrono_sensor/rtkernels/ray_utils.h"

using namespace optix;

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, root_node, , );
rtDeclareVariable(float, default_depth, , );
rtDeclareVariable(float3, default_color, , );

rtBuffer<float, 2> output_buffer;

// This kernel is launched once for each pixel in the image
RT_PROGRAM void depth_trace_camera() {
    size_t2 screen = output_buffer.size();

    // set the ray direction based on the proportion of image the pixel is located at
    float2 d = (make_float2(launch_index) + make_float2(0.5, 0.5)) / make_float2(screen) * 2.f - 1.f;
    // origin of the camera is  0,0,0 for now
    float3 ray_origin = {0, 0, 0};
    float3 ray_direction =
        normalize(float3{1.0, -d.x, d.y});  // flip y value to have image oriented with 0,0 in lower left

    // create a ray based on the calculated parameters
    optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon);

    // set the ray pay load
    PerRayData_radiance prd_radiance;
    prd_radiance.importance = 1.f;
    prd_radiance.depth = 0;
    prd_radiance.type = DEPTH;

    // launch the ray
    rtTrace(root_node, ray, prd_radiance);

    // set the output buffer to be what is returned in the payload
    output_buffer[launch_index] = prd_radiance.distance;
}

RT_PROGRAM void miss_function() {
    if (prd_radiance.type == DEPTH) {
        prd_radiance.distance = default_depth;
    } else if (prd_radiance.type == LIDAR) {
    } else if (prd_radiance.type == CAMERA) {
        prd_radiance.color = default_color;
    }
}
