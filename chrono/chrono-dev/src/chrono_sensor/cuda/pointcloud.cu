#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include "pointcloud.cuh"

namespace chrono {
namespace sensor {

// Converts 32bpp ARGB imgIn pixels to 8bpp Grayscale imgOut pixels
__global__ void pointcloud_from_depth_kernel(float* imgIn, float* imgOut, int numPixels, LidarParams params) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < numPixels) {
        int hIndex = index % params.horizontal_samples;
        int vIndex = index / params.horizontal_samples;

        float vAngle = (vIndex / (float)(params.vertical_samples)) * params.vFOV - params.vFOV / 2.;
        float hAngle = (hIndex / (float)(params.horizontal_samples)) * params.hFOV - params.hFOV / 2.;

        float range = imgIn[2 * index];

        float proj_xy = range * cos(vAngle);

        float x = proj_xy * cos(hAngle);
        float y = proj_xy * sin(hAngle);
        float z = range * sin(vAngle);
        imgOut[4 * index] = x;
        imgOut[4 * index + 1] = y;
        imgOut[4 * index + 2] = z;
        imgOut[4 * index + 3] = imgIn[2 * index + 1];
    }
}

void cuda_pointcloud_from_depth(void* bufDI, void* bufOut, int width, int height, LidarParams params) {
    int numPixels = width * height;
    const int nThreads = 512;
    int nBlocks = (numPixels + nThreads - 1) / nThreads;

    pointcloud_from_depth_kernel<<<nBlocks, nThreads>>>((float*)bufDI, (float*)bufOut, numPixels, params);
}

}  // namespace sensor
}  // namespace chrono
